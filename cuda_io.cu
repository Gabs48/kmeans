#include "hip/hip_runtime.h"
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
/*   File:         file_io.c                                                 */
/*   Description:  This program reads point data from a file                 */
/*                 and write cluster output to files                         */
/*   Input file format:                                                      */
/*                 ascii  file: each line contains 1 data object             */
/*                 binary file: first 4-byte integer is the number of data   */
/*                 objects and 2nd integer is the no. of features (or        */
/*                 coordinates) of each object                               */
/*                                                                           */
/*   Author:  Wei-keng Liao                                                  */
/*            ECE Department Northwestern University                         */
/*            email: wkliao@ece.northwestern.edu                             */
/*   Copyright, 2005, Wei-keng Liao                                          */
/*                                                                           */
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>     /* strtok() */
#include <sys/types.h>  /* open() */
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>     /* read(), close() */

#include "kmeans.h"

#define MAX_CHAR_PER_LINE 128

int infile_b;
FILE *infile_t;
int   lineLen;


/*---< file_read_head() >---------------------------------------------------------*/
int file_read_head(int   isBinaryFile,  /* flag: 0 or 1 */
                  char *filename,      /* input file name */
                  int  *numObjs,       /* no. data objects (local) */
                  int  *numCoords)     /* no. coordinates */
{
	int     len;
	ssize_t numBytesRead;

	if (isBinaryFile) {  /* input file is in raw binary format -------------*/
		if ((infile_b = open(filename, O_RDONLY, "0600")) == -1) {
			fprintf(stderr, "[file io] error: no such file (%s)\n", filename);
			return 0;
		}
		numBytesRead = read(infile_b, numObjs,    sizeof(int));
		assert(numBytesRead == sizeof(int));
		numBytesRead = read(infile_b, numCoords, sizeof(int));
		assert(numBytesRead == sizeof(int));
		if (_debug) {
			printf("[file io] file %s numObjs   = %d\n",filename,*numObjs);
			printf("[file io] file %s numCoords = %d\n",filename,*numCoords);
		}
        
	} else {  /* input file is in ASCII format -------------------------------*/
		char *line, *ret;

		if ((infile_t = fopen(filename, "r")) == NULL) {
			fprintf(stderr, "[file io] error: no such file (%s)\n", filename);
			return 0;
		}

		/* first find the number of objects */
		lineLen = MAX_CHAR_PER_LINE;
		line = (char*) malloc(lineLen);
		assert(line != NULL);

		(*numObjs) = 0;

		while (fgets(line, lineLen, infile_t) != NULL) {
			/* check each line to find the max line length */
			while (strlen(line) == lineLen-1) {
				/* this line read is not complete */
				len = strlen(line);
				fseek(infile_t, -len, SEEK_CUR);

				/* increase lineLen */
				lineLen += MAX_CHAR_PER_LINE;
				line = (char*) realloc(line, lineLen);
				assert(line != NULL);

				ret = fgets(line, lineLen, infile_t);
				assert(ret != NULL);
			}

			if (strtok(line, " \t\n") != 0)
				(*numObjs)++;
		}
		rewind(infile_t);
		if (_debug) printf("[file io] lineLen = %d\n",lineLen);

		/* find the no. objects of each object */
		(*numCoords) = 0;
		while (fgets(line, lineLen, infile_t) != NULL) {
			if (strtok(line, " \t\n") != 0) {
				/* ignore the id (first coordiinate): numCoords = 1; */
				while (strtok(NULL, " ,\t\n") != NULL) (*numCoords)++;
				break; /* this makes read from 1st object */
			}
		}
		rewind(infile_t);
		if (_debug) {
			printf("[file io] file %s numObjs   = %d\n",filename,*numObjs);
			printf("[file io] file %s numCoords = %d\n",filename,*numCoords);
		}
    }    
	return 1;
}

/*---< file_read_head() >---------------------------------------------------------*/
float** file_read_block(int   isBinaryFile,  /* flag: 0 or 1 */
                  char *filename,      /* input file name */
                  int  numObjs,       /* no. data objects (local) */
                  int  numCoords)     /* no. coordinates */
{
	int     i, j;
	ssize_t numBytesRead;
	float **objects;
    
	int len = numObjs * numCoords;
	
	if (_debug)
		printf("[file io] read a block of %ix%i objects\n", numObjs, numCoords);
	if (isBinaryFile) {  /* input file is in raw binary format -------------*/
		
		objects    = (float**)malloc(numObjs * sizeof(float*));
		assert(objects != NULL);
		objects[0] = (float*) malloc(len * sizeof(float));
        assert(objects[0] != NULL);
        for (i=1; i<numObjs; i++)
            objects[i] = objects[i-1] + numCoords;

		numBytesRead = read(infile_b, objects[0], len*sizeof(float));
		assert(numBytesRead == len*sizeof(float));

	} else {  /* input file is in ASCII format -------------------------------*/

		char *line = (char*) malloc(lineLen);
		int llen;
        objects    = (float**)malloc(numObjs * sizeof(float*));
        assert(objects != NULL);
        objects[0] = (float*) malloc(len * sizeof(float));
        assert(objects[0] != NULL);
        for (i=1; i<numObjs; i++)
            objects[i] = objects[i-1] + numCoords;

        i = 0;
        /* read all objects */
        while (fgets(line, lineLen, infile_t) != NULL && i < numObjs) {
			if (i == 0)
				llen = strlen(line);
            if (strtok(line, " \t\n") == NULL)
				continue;
            for (j=0; j<numCoords; j++)
                objects[i][j] = atof(strtok(NULL, " ,\t\n"));
            i++;
        }
		fseek(infile_t, -llen, SEEK_CUR);
        free(line);
    }
    //if(_debug)
		//printf("[file io] first data values: %.4f %.4f %.4f %.4f\n", objects[0][0], objects[0][1], objects[1][0], objects[1][1]);
	
    return objects;
}

       
int file_read_close(int isBinaryFile)
{
	if (isBinaryFile)
		close(infile_b);
	else
		fclose(infile_t);
	
	return 1;
}

/*---< file_write() >---------------------------------------------------------*/
int file_write(char      *filename,     /* input file name */
               int        numClusters,  /* no. clusters */
               int        numObjs,      /* no. data objects */
               int        numCoords,    /* no. coordinates (local) */
               float    **clusters,     /* [numClusters][numCoords] centers */
               int       *membership)   /* [numObjs] */
{
    FILE *fptr;
    int   i, j;
    char  outFileName[1024];

    /* output: the coordinates of the cluster centres ----------------------*/
    sprintf(outFileName, "%s.cluster_centres", filename);
    printf("\n[file io] writing coordinates of K=%d cluster centers to file \"%s\"\n",
           numClusters, outFileName);
    fptr = fopen(outFileName, "w");
    for (i=0; i<numClusters; i++) {
        fprintf(fptr, "%d ", i);
        for (j=0; j<numCoords; j++)
            fprintf(fptr, "%f ", clusters[i][j]);
        fprintf(fptr, "\n");
    }
    fclose(fptr);

    /* output: the closest cluster centre to each of the data points --------*/
    sprintf(outFileName, "%s.membership", filename);
    printf("[file io] writing membership of N=%d data objects to file \"%s\"\n",
           numObjs, outFileName);
    fptr = fopen(outFileName, "w");
    for (i=0; i<numObjs; i++)
        fprintf(fptr, "%d %d\n", i, membership[i]);
    fclose(fptr);

    return 1;
}
