
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
// #include <graphics.h>
// #include <X11/Xlib.h>

#	define W 400
#	define H 400

// FOR FUNCTION TESTS
// int main()
// {
// 	int nClusters = 50;
// 	int nObjects = 500;
// 	int i, j;
// 	float *xObjects, *yObjects, *xClusters, *yClusters;
//	int *membership;
// 	xObjects = (float*)malloc(nObjects * sizeof(float));
// 	yObjects = (float*)malloc(nObjects * sizeof(float));
// 	xClusters = (float*)malloc(nClusters * sizeof(float));
// 	yClusters = (float*)malloc(nClusters * sizeof(float));
//  membership = (int*)malloc(nObjects * sizeof(int));
// 	printf("Objects: ");
// 	for (i = 0; i < nObjects; i++) {
// 		xObjects[i] = (float)rand()/(float)(RAND_MAX/8);
// 		yObjects[i] = (float)rand()/(float)(RAND_MAX/30);
//		membership[i] = rand()/(RAND_MAX/nClusters);
// 		printf("(%.2f;", xObjects[i]);
// 		printf("%.2f;", yObjects[i]);
//		printf("%i) ", membership[i]);
// 	}
// 	printf("\nClusters: ");
// 	for (i = 0; i < nClusters; i++) {
// 		xClusters[i] = (float)rand()/(float)(RAND_MAX/10);
// 		yClusters[i] = (float)rand()/(float)(RAND_MAX/40);
// 		printf("(%.2f;", xClusters[i]);
// 		printf("%.2f) ", yClusters[i]);
// 	}
// 	graph_kmean(xObjects, yObjects, nObjects, xClusters, yClusters, nClusters, membership);
// }

void pdf_kmean(float* xObjects, float* yObjects, int nObjects, float* xClusters, float* yClusters, int nClusters, int* membership)
{
	int i, j;
	double min_x, max_x, min_y, max_y, scale, cx, cy;
	double *colors = (double*)malloc(sizeof(double) * nClusters * 3);
 
	// Open file
	FILE *fptr = fopen("results.eps", "w");
	if (fptr == NULL) {
		printf("[display] Cannot create results.pdf file!\n");
		return;
	}
	
	// Compute colors
	for (i = 0; i < nClusters; i++) {
		colors[3*i + 0] = (3 * (i + 1) % 11)/11.;
		colors[3*i + 1] = (7 * i % 11)/11.;
		colors[3*i + 2] = (9 * i % 11)/11.;
	}
 
	// Find max, min and range values
	max_x = max_y = -(min_x = min_y = HUGE_VAL);
	for (j = 0; j < nObjects; j++) {
		if (max_x < xObjects[j]) max_x = xObjects[j];
		if (min_x > xObjects[j]) min_x = xObjects[j];
		if (max_y < yObjects[j]) max_y = yObjects[j];
		if (min_y > yObjects[j]) min_y = yObjects[j];
	}
	scale = W / (max_x - min_x);
	if (scale > H / (max_y - min_y)) scale = H / (max_y - min_y);
	cx = (max_x + min_x) / 2;
	cy = (max_y + min_y) / 2;
	
	// Fill the PDF
	fprintf(fptr, "%%!PS-Adobe-3.0\n%%%%BoundingBox: -5 -5 %d %d\n", W + 10, H + 10);
	fprintf(fptr,  "/l {rlineto} def /m {rmoveto} def\n"
		"/c { .25 sub exch .25 sub exch .5 0 360 arc fill } def\n"
		"/s { moveto -2 0 m 2 2 l 2 -2 l -2 -2 l closepath "
		"	gsave 1 setgray fill grestore gsave 3 setlinewidth"
		" 1 setgray stroke grestore 0 setgray stroke }def\n"
	);
	for (i = 0; i < nClusters; i++) {
		fprintf(fptr, "%g %g %g setrgbcolor\n",
			colors[3*i], colors[3*i + 1], colors[3*i + 2]);
		for (j = 0; j < nObjects; j++) {
			if (membership[j] != i) continue;
			fprintf(fptr, "%.3f %.3f c\n",
				(xObjects[j] - cx) * scale + W / 2,
				(yObjects[j] - cy) * scale + H / 2);
		}
		fprintf(fptr, "\n0 setgray %g %g s\n",
			(xClusters[i] - cx) * scale + W / 2,
			(yClusters[i] - cy) * scale + H / 2);
	}
	fprintf(fptr, "\n%%%%EOF");
	fclose(fptr);
	free(colors);
}


void gui_kmean(float* xObjects, float* yObjects, int nObjects, float* xClusters, float* yClusters, int nClusters, int* membership)
{
// 	const int MIN_X = 3, MIN_Y = 3, RANGE_X = 634, RANGE_Y = 474; //, MAX_X = 637, MAX_Y = 477;
// 	int i;
// 	
// 	// Find max and min coordinates
// 	float xMax = -1000000, xMin = 1000000, yMax = -1000000, yMin = 1000000;
// 	for (i = 0; i < nObjects; i++) {
// 		if (xObjects[i] > xMax)
// 			xMax = xObjects[i];
// 		if (xObjects[i] < xMin)
// 			xMin = xObjects[i];
// 		if (yObjects[i] > yMax)
// 			yMax = yObjects[i];
// 		if (yObjects[i] < yMin)
// 			yMin = yObjects[i];
// 	}
// 	for (i = 0; i < nClusters; i++) {
// 		if (xClusters[i] > xMax)
// 			xMax = xClusters[i];
// 		if (xClusters[i] < xMin)
// 			xMin = xClusters[i];
// 		if (yClusters[i] > yMax)
// 			yMax = yClusters[i];
// 		if (yClusters[i] < yMin)
// 			yMin = yClusters[i];
// 	}
// 	
// 	// Rescale
// 	float xRange = xMax - xMin;
// 	float yRange = yMax - yMin;
// 	//printf("\nObjects: ");
// 	for (i = 0; i < nObjects; i++) {
// 		xObjects[i] = MIN_X + (xObjects[i] - xMin) * RANGE_X / xRange;
// 		yObjects[i] = MIN_Y + (yObjects[i] - yMin) * RANGE_Y / yRange;
// 		//printf("(%.2f;", xObjects[i]);
// 		//printf("%.2f) ", yObjects[i]);
// 	}
// 	//printf("\nClusters: ");
// 	for (i = 0; i < nClusters; i++) {
// 		xClusters[i] = MIN_X + (xClusters[i] - xMin) * RANGE_X / xRange;
// 		yClusters[i] = MIN_Y + (yClusters[i] - yMin) * RANGE_Y / yRange;
// 		//printf("(%.2f;", xClusters[i]);
// 		//printf("%.2f) ", yClusters[i]);
// 	}
// 	//printf("\n");
// 	
// 	// Open window
// 	XInitThreads();
// 	int gd = DETECT, gm;
// 	initgraph(&gd, &gm, NULL);
// 	
// 	// Display points
// 	int j;
// 	for (i = 0; i < nObjects; i++) {
// 		j = membership[i]%15;
// 		if (j==0) {
// 			j=14;
// 		} else if (membership[i] == 0) {
// 			j=15;
// 		}
// 		setcolor(j);
// 		pieslice(xObjects[i], yObjects[i], 0, 360, 1);
// 	}
// // 	for (i = 0; i < nClusters; i++){
// // 		j = i%15;
// // 		if (j==0)
// // 			j=2;
// // 		setcolor(j);
// // 		pieslice(xClusters[i], yClusters[i], 0, 360, 3);
// // 	}
// 	
// 	// Wait for any char as exit command 
// 	getch();
// 	closegraph();
	return;
}