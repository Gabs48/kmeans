#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <graphics.h>
#include <X11/Xlib.h>

// FOR FUNCTION TESTS
// int main()
// {
// 	int nClusters = 50;
// 	int nObjects = 500;
// 	int i, j;
// 	float *xObjects, *yObjects, *xClusters, *yClusters;
// 	xObjects = (float*)malloc(nObjects * sizeof(float));
// 	yObjects = (float*)malloc(nObjects * sizeof(float));
// 	xClusters = (float*)malloc(nClusters * sizeof(float));
// 	yClusters = (float*)malloc(nClusters * sizeof(float));
// 	printf("Objects: ");
// 	for (i = 0; i < nObjects; i++) {
// 		xObjects[i] = (float)rand()/(float)(RAND_MAX/8);
// 		yObjects[i] = (float)rand()/(float)(RAND_MAX/30);
// 		printf("(%.2f;", xObjects[i]);
// 		printf("%.2f) ", yObjects[i]);
// 	}
// 	printf("\nClusters: ");
// 	for (i = 0; i < nClusters; i++) {
// 		xClusters[i] = (float)rand()/(float)(RAND_MAX/10);
// 		yClusters[i] = (float)rand()/(float)(RAND_MAX/40);
// 		printf("(%.2f;", xClusters[i]);
// 		printf("%.2f) ", yClusters[i]);
// 	}
// 	graph_kmean(xObjects, yObjects, nObjects, xClusters, yClusters, nClusters);
// }

int graph_kmean(float* xObjects, float* yObjects, int nObjects, float* xClusters, float* yClusters, int nClusters, int* membership, float totalDistance)
{
	const int MIN_X = 3, MIN_Y = 3, MAX_X = 637, MAX_Y = 477, RANGE_X = 634, RANGE_Y = 474;
	int i;
	
	// Find max and min coordinates
	float xMax = -1000000, xMin = 1000000, yMax = -1000000, yMin = 1000000;
	for (i = 0; i < nObjects; i++) {
		if (xObjects[i] > xMax)
			xMax = xObjects[i];
		if (xObjects[i] < xMin)
			xMin = xObjects[i];
		if (yObjects[i] > yMax)
			yMax = yObjects[i];
		if (yObjects[i] < yMin)
			yMin = yObjects[i];
	}
	for (i = 0; i < nClusters; i++) {
		if (xClusters[i] > xMax)
			xMax = xClusters[i];
		if (xClusters[i] < xMin)
			xMin = xClusters[i];
		if (yClusters[i] > yMax)
			yMax = yClusters[i];
		if (yClusters[i] < yMin)
			yMin = yClusters[i];
	}
	//printf("\nxMax: %.2f; yMax: %.2f; xMin: %.2f; yMin: %.2f", xMax, yMax, xMin, yMin);
	
	// Rescale
	float xRange = xMax - xMin;
	float yRange = yMax - yMin;
	//printf("\nObjects: ");
	for (i = 0; i < nObjects; i++) {
		xObjects[i] = MIN_X + (xObjects[i] - xMin) * RANGE_X / xRange;
		yObjects[i] = MIN_Y + (yObjects[i] - yMin) * RANGE_Y / yRange;
		//printf("(%.2f;", xObjects[i]);
		//printf("%.2f) ", yObjects[i]);
	}
	//printf("\nClusters: ");
	for (i = 0; i < nClusters; i++) {
		xClusters[i] = MIN_X + (xClusters[i] - xMin) * RANGE_X / xRange;
		yClusters[i] = MIN_Y + (yClusters[i] - yMin) * RANGE_Y / yRange;
		//printf("(%.2f;", xClusters[i]);
		//printf("%.2f) ", yClusters[i]);
	}
	//printf("\n");
	
	// Open window
	XInitThreads();
	int gd = DETECT, gm;
	initgraph(&gd, &gm, NULL);
	
	// Display points
	int j;
	for (i = 0; i < nObjects; i++) {
		j = membership[i]%15;
		if (j==0) {
			j=14;
		} else if (membership[i] == 0) {
			j=15;
		}
		setcolor(j);
		pieslice(xObjects[i], yObjects[i], 0, 360, 1);
	}
// 	for (i = 0; i < nClusters; i++){
// 		j = i%15;
// 		if (j==0)
// 			j=2;
// 		setcolor(j);
// 		pieslice(xClusters[i], yClusters[i], 0, 360, 3);
// 	}
	printf("\nTotal distance: %.2f", totalDistance);
	
	// Wait for any char as exit command 
	getch();
	closegraph();
	return 0;
}